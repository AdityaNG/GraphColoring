#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string.h>
#include <vector>
#include <math.h>
#include <cassert>
#include <bits/stdc++.h> // std::reverse

#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

/*
TODO:
1. Remove C++ STL dependency in all functions, migrate to normal arrays
  1.1 (you may choose to migrate to migrate to CUDA's Thrust library)
2. DONE Remove any use of C++ strings, convert to c strings
3. DONE Remove any memory operations (malloc, free) from all the functions but main()
4. Rewrite the for loop in the main() fn as a kernel function
  4.1 Split up the task across the kernels
  4.2 Refer to make BruteForcePThread's handleDataArg() for details
5. Allocate memory on the GPU in the main function
6. Call the CUDA kernels appropriately

Useful:
1. To Compile:
  make BruteForceCUDA
2. To run:
  ./test.sh
*/

#define LOG_DATA false

using ll = long long;


std::vector<std::vector <ll>> graph;
ll* cgraph = &graph[][0];

// helper functions
std::string decToBase(ll num, ll base) {
  if(num==0)
    return "0";

  std::string base_num = "";
  while (num>0) {
    ll dig = ll(num%base);
    if(dig<10) {
      base_num += std::to_string(dig);
    } else {
      base_num += std::string(1, char('A'+dig-10));  // Using uppercase letters
    }
    num = num/base;
  }
  std::reverse(base_num.begin(), base_num.end());// To reverse the string
  return base_num;
}

std::string padZeros(std::string s, ll n) {
  while (s.length() < n) {
    s = "0" + s;
  }
  return s;
}

ll getLength(ll v) {
  ll l = 0;
  for (ll i=1; i<=v; i++) {
    l += pow(i, v);
  }
  return l;
}

std::pair<ll,ll> getItem(ll v, ll key) {
  ll l = getLength(v);
  assert(key <= l);

  key+=1;
  ll t = 1;
  ll c = 1;
  while (key>t) {
    key-=t;
    c+=1;
    t = pow(c, v);
  }
        
  key -= 1;

  return std::make_pair(key, c);
}

// coloring
__global__ void isSafeToColor(ll* cgraph, ll* color, int size, bool* answer) {
  ll V = size; 
  int xindex = blockIdx.x * blockDim.x + threadIdx.x;
  int yindex = xindex+1; //blockIdx.y * blockDim.y + threadIdx.y;
  //for (ll i = 0; i < V; i++)
    //for (ll j = i + 1; j < V; j++)
  if(xindex<V && yindex<V)
  {
    if (cgraph[xindex][yindex] == 1 && color[yindex] == color[xindex])
          *answer= false;
    *answer= true;
  }
}

int main(int argc, char ** argv) {

  ll v;
  std::cin >> v;

  // set to 0
  for (ll i=0; i<v; i++) {
    for (ll j=0; j<v; j++) {
      cgraph[i][j] = 0;
    }
  }

  // read input graph
  for (ll i=0; i<v; i++) {
    for (ll j=0; j<v; j++) {
      std::cin >> cgraph[i][j];
    }
  }

  ll min_count = v;

  ll l = getLength(v);
  for (ll i=0; i<l; i++) {
    //ll id = i * MAXTHREADS + (thread_number%MAXTHREADS);
    ll id = i;
    std::pair<ll,ll> c_j = getItem(v, id);
    ll c = c_j.second;
    ll j = c_j.first;
    
    //if (LOG_DATA) printf("(%lld, %lld)\n", c, j);
    if (c < min_count) {
      std::string col = padZeros(decToBase(j, c), v);
      
      if (LOG_DATA) printf("(%lld, %lld)\t->\t%s\n", c, j, col.c_str());

      // set to 0
      ll* colors[v];
      for (ll i=0; i<v; i++) {
        colors[i] = 0;
      }
      
      // set colors
      for (ll t=0; t<v; t++) {
        colors[t] = col.at(t) - '0';
      }

      assert(col.length()==v);
      // call kernel function
      bool h_answer;
      bool* d_answer;
      
      ll* d_graph;
      ll* d_colors;
      
      hipMalloc((void **)&d_graph, v*v*sizeof(ll));
      hipMalloc((void **)&d_colors, v*sizeof(ll));
      hipMalloc(&d_answer, sizeof(bool));

      hipMemcpy(d_graph, &cgraph, v*v*sizeof(ll), hipMemcpyHostToDevice);
      hipMemcpy(d_colors, &colors, v*sizeof(ll), hipMemcpyHostToDevice);
      isSafeToColor<<<v*v,1>>>(d_graph, d_colors, v, d_answer);
      hipMemcpy(&h_answer, d_answer, sizeof(bool), hipMemcpyDeviceToHost); 
      
      if (h_answer) {
        min_count = c;
      }
      hipFree(d_answer);
      hipFree(d_graph); 
      hipFree(d_colors); 
    }
  }

  //if (LOG_DATA)
  printf("min=%lld\n", min_count);

}
