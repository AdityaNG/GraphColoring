#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string.h>
#include <vector>
#include <math.h>
#include <cassert>
#include <bits/stdc++.h> // std::reverse

#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

/*
TODO:
1. Remove C++ STL dependency in all functions, migrate to normal arrays
  1.1 (you may choose to migrate to migrate to CUDA's Thrust library)
2. Remove any use of C++ strings, convert to c strings
3. Remove any memory operations (malloc, free) from all the functions but main()
4. Rewrite the for loop in the main() fn as a kernel function
  4.1 Split up the task across the kernels
  4.2 Refer to make BruteForcePThread's handleDataArg() for details
5. Allocate memory on the GPU in the main function
6. Call the CUDA kernels appropriately

Useful:
1. To Compile:
  make BruteForceCUDA
2. To run:
  ./test.sh
*/

#define LOG_DATA false

using ll = long long;

std::vector<std::vector <ll>> graph;

std::string decToBase(ll num, ll base) {
  if(num==0)
    return "0";

  std::string base_num = "";
  while (num>0) {
    ll dig = ll(num%base);
    if(dig<10) {
      base_num += std::to_string(dig);
    } else {
      base_num += std::string(1, char('A'+dig-10));  // Using uppercase letters
    }
    num = num/base;
  }
  std::reverse(base_num.begin(), base_num.end());// To reverse the string
  return base_num;
}

std::string padZeros(std::string s, ll n) {
  while (s.length() < n) {
    s = "0" + s;
  }
  return s;
}

ll getLength(ll v) {
  ll l = 0;
  for (ll i=1; i<=v; i++) {
    l += pow(i, v);
  }
  return l;
}

std::pair<ll,ll> getItem(ll v, ll key) {
  ll l = getLength(v);
  assert(key <= l);

  key+=1;
  ll t = 1;
  ll c = 1;
  while (key>t) {
    key-=t;
    c+=1;
    t = pow(c, v);
  }
        
  key -= 1;

  return std::make_pair(key, c);
}

bool isSafeToColor(std::vector<std::vector<ll>> graph, std::vector <ll> color) {
  ll V = color.size();
  for (ll i = 0; i < V; i++)
    for (ll j = i + 1; j < V; j++)
      if (graph[i][j] == 1 && color[j] == color[i])
        return false;
  return true;
}

int main(int argc, char ** argv) {

  ll v;
  std::cin >> v;

  graph = std::vector<std::vector <ll>>(v, std::vector<ll>(v, 0));

  for (ll i=0; i<v; i++) {
    for (ll j=0; j<v; j++) {
      std::cin >> graph[i][j];
    }
  }

  ll min_count = v;

  ll l = getLength(v);
  for (ll i=0; i<l; i++) {
    //ll id = i * MAXTHREADS + (thread_number%MAXTHREADS);
    ll id = i;
    std::pair<ll,ll> c_j = getItem(v, id);
    ll c = c_j.second;
    ll j = c_j.first;
    
    //if (LOG_DATA) printf("(%lld, %lld)\n", c, j);
    if (c < min_count) {
      std::string col = padZeros(decToBase(j, c), v);
      
      if (LOG_DATA) printf("(%lld, %lld)\t->\t%s\n", c, j, col.c_str());

      std::vector<ll> colors(v, 0);
      for (ll t=0; t<v; t++) {
        colors[t] = col.at(t) - '0';
      }

      assert(col.length()==v);
      if (isSafeToColor(graph, colors)) {
        min_count = c;
      }
    }
  }

  //if (LOG_DATA)
  printf("min=%lld\n", min_count);

}